#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 9

// Statically defined device memory
__device__ int d_a[N];
__device__ int d_b[N];
__device__ int d_c[N];

__global__ void add() {
    int i = threadIdx.x;
    d_c[i] = d_a[i] + d_b[i];
}

int main() {
    int a[N] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
    int b[N] = {10, 20, 30, 40, 50, 60, 70, 80, 90};
    int c[N] = {0};
    
    // Copy data to GPU using cudaMemcpyToSymbol
    hipMemcpyToSymbol(HIP_SYMBOL(d_a), a, N * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_b), b, N * sizeof(int));
    
    // Run kernel with 1 block of N threads
    add<<<1, N>>>();
    
    // Check for kernel errors and synchronize
    hipGetLastError();
    hipDeviceSynchronize();
    
    // Copy result back using cudaMemcpyFromSymbol
    hipMemcpyFromSymbol(c, HIP_SYMBOL(d_c), N * sizeof(int));
    
    // Print results
    printf("Results:\n");
    for (int i = 0; i < N; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }
    
    return 0;
}
