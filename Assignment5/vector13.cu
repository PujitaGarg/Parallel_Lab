#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 9

// Statically defined device memory
__device__ int d_a[N];
__device__ int d_b[N];
__device__ int d_c[N];

__global__ void add() {
    int i = threadIdx.x;
    d_c[i] = d_a[i] + d_b[i];
}

int main() {
    int a[N] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
    int b[N] = {10, 20, 30, 40, 50, 60, 70, 80, 90};
    int c[N] = {0};
    
    // Query device properties
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);  // Get properties of device 0
    
    // Calculate theoretical bandwidth
    double memoryClockRateGHz = prop.memoryClockRate * 1e-6;  // Convert kHz to GHz
    double memoryBusWidthBits = prop.memoryBusWidth;
    
    // Theoretical bandwidth in GB/s (accounting for DDR being double pumped)
    double theoreticalBW = (memoryClockRateGHz * memoryBusWidthBits * 2) / 8;
    
    // Print device information and bandwidth
    printf("Device: %s\n", prop.name);
    printf("Memory Clock Rate: %.0f MHz\n", prop.memoryClockRate * 1e-3);
    printf("Memory Bus Width: %d bits\n", prop.memoryBusWidth);
    printf("Theoretical Memory Bandwidth: %.2f GB/s\n\n", theoreticalBW);
    
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    
    // Copy data to GPU using hipMemcpyToSymbol
    hipMemcpyToSymbol(HIP_SYMBOL(d_a), a, N * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_b), b, N * sizeof(int));
    
    // Record start event
    hipEventRecord(start);
    
    // Run kernel with 1 block of N threads
    add<<<1, N>>>();
    
    // Record stop event
    hipEventRecord(stop);
    
    // Check for kernel errors and synchronize
    hipGetLastError();
    hipDeviceSynchronize();
    
    // Calculate elapsed time
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // Copy result back using hipMemcpyFromSymbol
    hipMemcpyFromSymbol(c, HIP_SYMBOL(d_c), N * sizeof(int));
    
    // Print results
    printf("Results:\n");
    for (int i = 0; i < N; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }
    
    // Print timing information
    printf("\nKernel execution time: %.3f milliseconds\n", milliseconds);
    
    // Clean up events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    return 0;
}
