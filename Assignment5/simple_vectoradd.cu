#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add(int *a, int *b, int *c) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main() {
    const int N = 9;
    int a[N] = {1, 2, 3, 4, 5,6,7,8,9};
    int b[N] = {10, 20, 30, 40, 50,60,70,80,90};
    int c[N] = {0};
    
    int *d_a, *d_b, *d_c;
    
    // Allocate GPU memory
    hipMalloc(&d_a, N * sizeof(int));
    hipMalloc(&d_b, N * sizeof(int));
    hipMalloc(&d_c, N * sizeof(int));
    
    // Copy data to GPU
    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);
    
    // Run kernel with 1 block of N threads
    add<<<1, N>>>(d_a, d_b, d_c);
    
    // Check for kernel errors and synchronize
    hipGetLastError();
    hipDeviceSynchronize();
    
    // Copy result back
    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);
    
    // Print results
    printf("Results:\n");
    for (int i = 0; i < N; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }
    
    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    return 0;
}
